
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

extern "C"
__global__ void piCalc(int *result)
{
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int sum = 0;
  unsigned int N = 1000; // samples per thread unsigned
  int seed = tid;
  hiprandState s; // seed a random number generator
  hiprand_init(seed, 0, 0, &s);
  // take N samples in a quarter circle
  for(unsigned int i = 0; i < N; ++i) {
    // draw a sample from the unit square
    float x = hiprand_uniform(&s);
    float y = hiprand_uniform(&s); // measure distance from the origin
    float dist = sqrtf(x*x + y*y);
    // add 1.0f if (u0,u1) is inside the quarter circle
    if(dist <= 1.0f) sum += 1;
  }
  result[tid] = sum;
}
